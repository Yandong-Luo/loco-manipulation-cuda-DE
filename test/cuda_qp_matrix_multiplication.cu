
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <cmath>
#include <sys/stat.h>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <memory>
#include <string.h>
#include <cstdint>

#define N 4    // 向量维度
#define COL 2  // 列数

__constant__ float Q[16] = {
    1.0f,  0.0f,  0.0f,  0.0f,
    0.0f, 50.0f,  0.0f,  0.0f,
    0.0f,  0.0f,  1.0f,  0.0f,
    0.0f,  0.0f,  0.0f, 50.0f
};

void printMatrix(float (*matrix)[4], int row) {
    for(int i = 0; i < row; i++) {
        std::cout << "[ ";
        for(int j = 0; j < 4; j++) {
            std::cout << matrix[i][j] << " ";
        }
        std::cout << "]" << std::endl;
    }
}

int main(void) {
    float alpha = 1.0;
    float beta = 0.0;
    
    // 初始化数据
    // float h_x[N][COL] = {{1,1},{2,2},{3,3},{4,4}};
    float h_x[2][4] = {{-0.023370, 0.1657, -0.618915, 5.1048},{1,2,3,4}};
    float h_Q[N][N] = {           // 示例对称矩阵
        {1, 0, 0, 0},
        {0, 1, 0, 0},
        {0, 0, 1, 0},
        {0, 0, 0, 1}
    };
    float h_temp[4][2] = {0};   // 临时存储 Qx 的结果
    float h_result[COL][COL] = {0}; // 2x2 结果矩阵

    // 分配设备内存
    float *d_x, *d_Q, *d_temp, *d_result;
    hipMalloc((void**)&d_x, N * COL * sizeof(float));
    hipMalloc((void**)&d_Q, N * N * sizeof(float));
    hipMalloc((void**)&d_temp, N * COL * sizeof(float));
    hipMalloc((void**)&d_result, COL * COL * sizeof(float));

    // 拷贝数据到设备
    hipMemcpy(d_x, h_x, N * COL * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Q, h_Q, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_temp, 0, N * COL * sizeof(float));

    // 创建 CUBLAS 句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // // 步骤1: Qx
    // cublasSgemm(handle,
    //     CUBLAS_OP_N, CUBLAS_OP_N,  // Q * x
    //     2, 4, 4,
    //     &alpha,
    //     d_x, 2,
    //     d_Q, 4,   // Q: N×N
    //     &beta,
    //     d_temp, 2 // Qx: N×COL
    // );

    // // 步骤2: x^T * (Qx)
    // cublasSgemm(handle,
    //     CUBLAS_OP_N, CUBLAS_OP_T,  // 注意这里的转置
    //     2, 2, 4,
    //     &alpha,
    //     d_x, 2,      
    //     d_temp, 2,   
    //     &beta,
    //     d_result, 2 
    // );
    // 第一个cublasSgemm: 计算 Q×x^T
    // 步骤1: Qx^T （注意这里h_x在cublas看来是4×2的矩阵）
    hipblasSgemm(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,  // x^T * Q 
        2, 4, 4,                   // m=2(x的列数), n=4(Q的列数), k=4(Q的行数)
        &alpha,
        d_x, 4,                    // x在cublas看来是4×2的，所以lda=4
        d_Q, 4,                    // Q是4×4矩阵
        &beta,
        d_temp, 2                  // 结果是2×4矩阵
    );

    // 步骤2: (x^TQ)x
    hipblasSgemm(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // temp * x
        2, 2, 4,                   // m=2(temp的行数), n=2(x的列数), k=4(temp的列数)
        &alpha,
        d_temp, 2,                 // temp是2×4矩阵
        d_x, 4,                    // x在cublas看来是4×2的矩阵
        &beta,
        d_result, 2                // 结果是2×2矩阵
    );
    // 拷贝结果回主机
    hipMemcpy(h_temp, d_temp, 4 * 2 * sizeof(float), hipMemcpyDeviceToHost);
    // 拷贝结果回主机
    hipMemcpy(h_result, d_result, COL * COL * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果
    std::cout << "X matrix:" << std::endl;
    printMatrix(h_x, 2);
    
    std::cout << "\nQ matrix:" << std::endl;
    for(int i = 0; i < N; i++) {
        std::cout << "[ ";
        for(int j = 0; j < N; j++) {
            std::cout << h_Q[i][j] << " ";
        }
        std::cout << "]" << std::endl;
    }
    
    std::cout << "\nResult matrix (2x2):" << std::endl;
    for(int i = 0; i < COL; i++) {
        std::cout << "[ ";
        for(int j = 0; j < COL; j++) {
            std::cout << h_result[i][j] << " ";
        }
        std::cout << "]" << std::endl;
    }

    std::cout << "\temp matrix (2x2):" << std::endl;
    for(int i = 0; i < 4; i++) {
        std::cout << "[ ";
        for(int j = 0; j < 2; j++) {
            std::cout << h_temp[i][j] << " ";
        }
        std::cout << "]" << std::endl;
    }

    // 清理资源
    hipFree(d_x);
    hipFree(d_Q);
    hipFree(d_temp);
    hipFree(d_result);
    hipblasDestroy(handle);

    return 0;
}