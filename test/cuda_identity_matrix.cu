
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <cmath>
#include <sys/stat.h>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <memory>
#include <string.h>
#include <cstdint>

#define MATRIX_SIZE 40  // 矩阵大小

__constant__ float Q[16] = {
    1.0f,  0.0f,  0.0f,  0.0f,
    0.0f, 50.0f,  0.0f,  0.0f,
    0.0f,  0.0f,  1.0f,  0.0f,
    0.0f,  0.0f,  0.0f, 50.0f
};

// 用于打印矩阵的辅助函数
void printMatrix(float* matrix, int size) {
    for(int i = 0; i < size; i++) {
        std::cout << "[ ";
        for(int j = 0; j < size; j++) {
            std::cout << matrix[i * size + j] << " ";
        }
        std::cout << "]" << std::endl;
    }
}

int main(void) {
    // 创建 CUBLAS 句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 在主机和设备上分配内存
    float* h_matrix = new float[MATRIX_SIZE * MATRIX_SIZE]();  // 初始化为0
    float* d_matrix;
    float* d_ones;
    
    // 为设备矩阵和单位向量分配内存
    hipMalloc((void**)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void**)&d_ones, MATRIX_SIZE * sizeof(float));
    
    // 将矩阵初始化为0
    hipMemset(d_matrix, 0, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    
    // 在CPU上创建一个全1向量
    float* h_ones = new float[MATRIX_SIZE];
    for(int i = 0; i < MATRIX_SIZE; i++) {
        h_ones[i] = 1.0f;
    }
    
    // 将全1向量复制到GPU
    hipMemcpy(d_ones, h_ones, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // 构建单位矩阵
    float alpha = 1.0f;
    // 使用cublasSaxpy设置对角线元素为1
    // incx为0表示使用相同的源值，incy为MATRIX_SIZE+1以跳到下一个对角线元素
    hipblasStatus_t status = hipblasSaxpy(handle, MATRIX_SIZE, &alpha, 
                                      d_ones, 1,  // 源向量是全1向量
                                      d_matrix, MATRIX_SIZE + 1);  // 步长为MATRIX_SIZE + 1
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSaxpy failed with error %d\n", status);
        return -1;
    }

    // 将结果拷贝回主机
    hipMemcpy(h_matrix, d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), 
               hipMemcpyDeviceToHost);

    // 打印结果矩阵
    std::cout << "\nIdentity matrix (" << MATRIX_SIZE << "x" << MATRIX_SIZE << "):" << std::endl;
    printMatrix(h_matrix, MATRIX_SIZE);

    // 验证结果
    bool isCorrect = true;
    for(int i = 0; i < MATRIX_SIZE; i++) {
        for(int j = 0; j < MATRIX_SIZE; j++) {
            float expected = (i == j) ? 1.0f : 0.0f;
            if (std::abs(h_matrix[i * MATRIX_SIZE + j] - expected) > 1e-6) {
                isCorrect = false;
                printf("Error at position (%d,%d): Expected %f, got %f\n", 
                       i, j, expected, h_matrix[i * MATRIX_SIZE + j]);
            }
        }
    }
    
    if (isCorrect) {
        std::cout << "\nIdentity matrix created successfully!" << std::endl;
    } else {
        std::cout << "\nError in creating identity matrix!" << std::endl;
    }

    // 清理资源
    delete[] h_matrix;
    delete[] h_ones;
    hipFree(d_matrix);
    hipFree(d_ones);
    hipblasDestroy(handle);

    return 0;
}