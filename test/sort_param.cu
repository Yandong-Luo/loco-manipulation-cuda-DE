#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_PARAM_MAX_SIZE 16
#define T 256  // template parameter for SortParamBasedBitonic

// Error checking macro
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(1); \
    } \
}

// Helper function to print arrays
void printArrays(float* fitness, float* params, int size) {
    printf("\nFitness values:\n");
    for(int i = 0; i < size; i++) {
        printf("%.2f ", fitness[i]);
    }
    // printf("\n\nParam values (first parameter only):\n");
    // for(int i = 0; i < size; i++) {
    //     printf("%.2f ", params[i * CUDA_PARAM_MAX_SIZE]);
    // }
    printf("\n\n");
}

__device__ __forceinline__ void BitonicWarpCompare(float &param, float &fitness, int lane_mask){
    float mapping_param = __shfl_xor_sync(0xffffffff, param, lane_mask);
    float mapping_fitness = __shfl_xor_sync(0xffffffff, fitness, lane_mask);
    // determine current sort order is increase (1.0) or decrease (-1.0)
    float sortOrder = (threadIdx.x > (threadIdx.x ^ lane_mask)) ? -1.0 : 1.0;

    if(sortOrder * (mapping_fitness - fitness) < 0.f){
        param = mapping_param;
        fitness = mapping_fitness;
    }
}

__global__ void SortParamBasedBitonic3(float *all_param, float *all_fitness){
    if (all_param == nullptr || all_fitness == nullptr) return;
    // if (threadIdx.x >= T)   return;
    // each block have a share memory
    __shared__ float sm_sorted_fitness[2*T];
    __shared__ float sm_sorted_param[2*T];
    int param_id = blockIdx.x;
    int sol_id = threadIdx.x;
    float current_param;
    float current_fitness;

    current_param = all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id];
    current_fitness = all_fitness[sol_id];
     

    int compare_idx;
    float mapping_param, mapping_fitness, sortOrder;

    // Sort the contents of 32 threads in a warp based on Bitonic merge sort. Implement detail is the alternative representation of https://en.wikipedia.org/wiki/Bitonic_sorter
    BitonicWarpCompare(current_param, current_fitness, 1);

    BitonicWarpCompare(current_param, current_fitness, 3);
    BitonicWarpCompare(current_param, current_fitness, 1);

    BitonicWarpCompare(current_param, current_fitness, 7);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    BitonicWarpCompare(current_param, current_fitness, 15);
    BitonicWarpCompare(current_param, current_fitness, 4);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    // above all finish the sorting 16 threads in Warp, continue to finish 2 group of 16 threads
    BitonicWarpCompare(current_param, current_fitness, 31);
    BitonicWarpCompare(current_param, current_fitness, 8);
    BitonicWarpCompare(current_param, current_fitness, 4);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    // above all finsh the sort for each warp, continue to finish the sort between different warp by share memory.
    // record the warp sorting result to share memory
    sm_sorted_param[sol_id ] = current_param;
    sm_sorted_fitness[sol_id] = current_fitness;
    
    // Wait for all thread finish above computation
    __syncthreads();

    if (T == 64){
        compare_idx = sol_id ^ 63;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];

        sortOrder = (threadIdx.x > (threadIdx.x ^ 63)) ? -1.0 : 1.0;

        if(sortOrder * (mapping_fitness - current_fitness) < 0.f){
            current_param = mapping_param;
            current_fitness = mapping_fitness;
        }
        
        // Wait for the sort between two warp finish
        __syncthreads();
        // Now, we can come back to the sorting in the warp
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);
    }
    else if(T == 128){
        sm_sorted_param[sol_id ] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
        compare_idx = threadIdx.x ^ 127;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;

        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
            sm_sorted_fitness[threadIdx.x] = current_fitness;
            sm_sorted_param[threadIdx.x] = current_param;
        }
    
        __syncthreads();
        
        compare_idx = threadIdx.x ^ 32;
        mapping_fitness = sm_sorted_fitness[compare_idx];
        mapping_param = sm_sorted_param[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;
        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
        }
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);
    }
    else if(T == 256){
        // 记录当前warp排序结果到共享内存
        sm_sorted_param[sol_id] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
        
        // 第一次大规模比较：与255异或（整体排序）
        compare_idx = threadIdx.x ^ 255;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;

        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
            sm_sorted_fitness[threadIdx.x] = current_fitness;
            sm_sorted_param[threadIdx.x] = current_param;
        }
        
        __syncthreads();
        
        // 第二次大规模比较：与64异或（128个元素分组排序）
        compare_idx = threadIdx.x ^ 64;
        mapping_fitness = sm_sorted_fitness[compare_idx];
        mapping_param = sm_sorted_param[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;
        
        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
            sm_sorted_fitness[threadIdx.x] = current_fitness;
            sm_sorted_param[threadIdx.x] = current_param;
        }
        
        __syncthreads();
        
        // 第三次大规模比较：与32异或（64个元素分组排序）
        compare_idx = threadIdx.x ^ 32;
        mapping_fitness = sm_sorted_fitness[compare_idx];
        mapping_param = sm_sorted_param[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;
        
        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
        }
        
        // 继续进行warp内部的比较排序
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);
    }

    if (blockIdx.x < CUDA_PARAM_MAX_SIZE){
        all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = current_param;
    }
    if (blockIdx.x == 0)    all_fitness[threadIdx.x] = current_fitness;
}

int main() {
    // Host arrays
    float *h_fitness, *h_params;
    // Device arrays
    float *d_fitness, *d_params;

    CHECK_CUDA(hipSetDevice(0));
    
    // Allocate host memory
    hipHostAlloc(&h_fitness, 2*T * sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&h_params, 2*T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipHostMallocDefault);
    // Initialize random seed
    srand(time(NULL));
    printf("T:%d\n", T);
    // Initialize fitness with decreasing values
    for(int i = 0; i < 2*T; i++) {
        h_fitness[i] = (float)(2*T - i);  // Creates values from T down to 1
    }
    
    // Initialize params with random values
    for(int i = 0; i < 2*T * CUDA_PARAM_MAX_SIZE; i++) {
        h_params[i] = (float)rand() / RAND_MAX * 100.0f;  // Random values between 0 and 100
    }
    
    printf("Initial arrays:");
    printArrays(h_fitness, h_params, 2*T);
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc(&d_fitness, 2*T * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_params, 2*T * CUDA_PARAM_MAX_SIZE * sizeof(float)));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_fitness, h_fitness, 2*T * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_params, h_params, 2*T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    // We need CUDA_PARAM_MAX_SIZE blocks because we're sorting each parameter independently
    // SortParamBasedBitonic<<<CUDA_PARAM_MAX_SIZE, T>>>(d_params, d_fitness);
    
    // SortParamBasedBitonic2<<<CUDA_PARAM_MAX_SIZE, 2*T>>>(d_params, d_fitness);

    SortParamBasedBitonic3<<<CUDA_PARAM_MAX_SIZE, 2*T>>>(d_params, d_fitness);

    
    // Check for kernel launch errors
    CHECK_CUDA(hipGetLastError());
    
    // Wait for GPU to finish
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_fitness, d_fitness, 2*T * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_params, d_params, 2*T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("Sorted arrays:");
    printArrays(h_fitness, h_params, 2*T);
    
    // Verify sorting
    bool sorted = true;
    for(int i = 1; i < 2*T; i++) {
        if(h_fitness[i-1] < h_fitness[i]) {
            sorted = false;
            printf("Error: Array not properly sorted at index %d\n", i);
            break;
        }
    }
    if(sorted) {
        printf("Verification: Arrays successfully sorted in descending order!\n");
    }
    
    // Cleanup
    CHECK_CUDA(hipHostFree(h_fitness));
    CHECK_CUDA(hipHostFree(h_params));
    CHECK_CUDA(hipFree(d_fitness));
    CHECK_CUDA(hipFree(d_params));
    
    return 0;
}